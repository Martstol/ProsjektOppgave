#include "hip/hip_runtime.h"
#include "PetscWind.cuh"
#include "CudaHelpers.cuh"

void setConstants(int dimx, int dimy, int dimz, int tdim) {
	// used in snow sim to convert snow particle
	// positions to wind grid space
	float temp[3] = {
	    (float)dimx / (float)SCENE_X,
	    (float)dimy / (float)SCENE_Y,
	    (float)dimz / (float)SCENE_Z
	};
	hipMemcpyToSymbol(HIP_SYMBOL(convert), temp, 3*sizeof(float), 
        0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(terrain_dim), &tdim, sizeof(int), 
        0, hipMemcpyHostToDevice);
}

void initGPUWindResources(hipArray **windVelArrayDevice, 
    int dimx, int dimy, int dimz, int tdim) {

    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    hipExtent extent = make_hipExtent(dimx, dimz, dimy);
    CUDA_SAFE_CALL(hipMalloc3DArray(windVelArrayDevice, &desc, extent));

    wind_vel_tex.filterMode = hipFilterModeLinear;
    wind_vel_tex.addressMode[0] = hipAddressModeClamp;
    wind_vel_tex.addressMode[1] = hipAddressModeClamp;
    wind_vel_tex.addressMode[2] = hipAddressModeClamp;
    CUDA_SAFE_CALL(hipBindTextureToArray(wind_vel_tex, *windVelArrayDevice));

	setConstants(dimx, dimy, dimz, tdim);
}

void windToGPU(float4 *windVel, hipArray *windVelArrayDevice, 
    int dimx, int dimy, int dimz) {

    hipMemcpy3DParms parm = {0};
    parm.srcPtr = make_hipPitchedPtr(windVel, dimx*sizeof(float4), dimx, dimz);
    parm.dstArray = windVelArrayDevice;
    parm.extent = make_hipExtent(dimx, dimz, dimy);
    parm.kind = hipMemcpyHostToDevice;
    CUDA_SAFE_CALL(hipMemcpy3D(&parm));
}

void freeGPUWindResources(hipArray *windVelArrayDevice) {
    CUDA_SAFE_CALL(hipUnbindTexture(wind_vel_tex));
    CUDA_SAFE_CALL(hipFreeArray(windVelArrayDevice));
}

void getTerrainMapFromGPU(float4 *terrainMap, int tdim) {
	float4* device_map = get_terrain_vertices();
	CUDA_SAFE_CALL(hipMemcpy(terrainMap, device_map, 
        sizeof(float4)*tdim*tdim, hipMemcpyDeviceToHost));
}